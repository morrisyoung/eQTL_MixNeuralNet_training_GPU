#include "hip/hip_runtime.h"
// the main optimization routine

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unordered_map>
#include <string>
#include <vector>
#include "basic.h"
#include <forward_list>
#include <utility>
#include "genotype.h"
#include "expression.h"
#include "optimization.h"
#include "global.h"
#include "main.h"  // typedef struct tuple_long
#include <math.h>       /* exp */
#include "opt_subroutine.h"
#include "opt_para_save.h"
#include "opt_debugger.h"
#include "lib_matrix.h"
#include "opt_hierarchy.h"
// includes CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#include <sys/time.h>
#include <time.h>       /* clock_t, clock, CLOCKS_PER_SEC */




using namespace std;




//====================================== local global variables ========================================
// these variables are specially designed for this routine -- optimization
// need to initialize some local containers:
array<float *, NUM_CHR> snp_dosage_list;
float * gene_rpkm_exp;  // with length "num_gene"
float * cellenv_hidden_var;  // with length "num_cellenv"
float * batch_var;  // with length "num_batch"
float * batch_hidden_var;  // with length "num_batch_hidden"


// parameter derivative containers:
vector<Matrix_imcomp> cube_para_dev_cis_gene;
Matrix matrix_para_dev_snp_cellenv;
vector<Matrix> cube_para_dev_cellenv_gene;
Matrix matrix_para_dev_batch_batch_hidden;
Matrix matrix_para_dev_batch_hidden_gene;




// GPU global variables (whenever it comes to d_xxx (GPU device memory), it's an array other than matrix)
//matrix_para_dev_snp_cellenv --> float * d_para_dev_snp_cellenv
float * d_para_snp_cellenv;
//vector<Matrix> cube_para_dev_cellenv_gene --> float * d_para_dev_cellenv_gene
vector<float *> d_list_para_cellenv_gene;
//Matrix matrix_para_dev_batch_batch_hidden --> float * d_para_dev_batch_batch_hidden
float * d_para_batch_batch_hidden;
//Matrix matrix_para_dev_batch_hidden_gene --> float * d_para_dev_batch_hidden_gene
float * d_para_batch_hidden_gene;


//matrix_para_dev_snp_cellenv --> float * d_para_dev_snp_cellenv
float * d_para_dev_snp_cellenv;
//vector<Matrix> cube_para_dev_cellenv_gene --> float * d_para_dev_cellenv_gene
vector<float *> d_list_para_dev_cellenv_gene;
//Matrix matrix_para_dev_batch_batch_hidden --> float * d_para_dev_batch_batch_hidden
float * d_para_dev_batch_batch_hidden;
//Matrix matrix_para_dev_batch_hidden_gene --> float * d_para_dev_batch_hidden_gene
float * d_para_dev_batch_hidden_gene;


// temp space for Matrix Multiplication
//matrix_para_dev_snp_cellenv --> float * d_para_dev_snp_cellenv
float * d_temp_snp_cellenv;
//vector<Matrix> cube_para_dev_cellenv_gene --> float * d_para_dev_cellenv_gene
float * d_temp_cellenv_gene;
//Matrix matrix_para_dev_batch_batch_hidden --> float * d_para_dev_batch_batch_hidden
float * d_temp_batch_batch_hidden;
//Matrix matrix_para_dev_batch_hidden_gene --> float * d_para_dev_batch_hidden_gene
float * d_temp_batch_hidden_gene;



// intermediate variables
int * d_etissue_index_p;
float * d_snp;						// real data
float * d_expr;						// real data
float * d_gene_rpkm_exp;  			// with length "num_gene"
float * d_gene_rpkm_exp_cellenv;  	// with length "num_gene"
float * d_gene_rpkm_exp_batch;  	// with length "num_gene"
float * d_error_list;				// with length "num_gene"
float * d_cellenv_hidden_var;  		// with length "num_cellenv"
float * d_batch_var;  				// with length "num_batch"
float * d_batch_hidden_var;  		// with length "num_batch_hidden"






//=====================================================
//********************** hierarchy ********************
//=====================================================
// containers:
vector<Matrix_imcomp> cube_para_cis_gene_parent;
vector<Matrix> cube_para_cellenv_gene_parent;


// the hierarchy:
// what to build for the hierarchy computing (prepared and renewed):
//		[2]. hashing all the leaves to their parents (in order to retrieve the parental variable array)
//		[4]. hashing all the internal nodes to their children and parent (with length to them, or the variance), in order to build the computational matrices
//		[5]. having a bi-directional list for the internal nodes (in order to build and fill in computtional matrices)
//		(6). building the tissue distance list (to its parent) based on etissue_list, to be used by the actual regularization
unordered_map<string, hierarchy_neighbor> hash_leaf_parent;		// --> [2]
unordered_map<string, vector< hierarchy_neighbor >> hash_internode_neighbor;
																// --> [4]
vector<string> internode_list;									// --> [5]
unordered_map<string, int> internode_index_map;					// --> [5]
int num_internode;
vector<float> etissue_dis_par_list;								// --> (6)





// some assistant components:
// the prior number for each un-pruned snp for regularization (from pruned snps and chromatin states); per etissue, per chromosome, for each snp
// TODO: we also still need to integrate distance prior later on with the following prior information
vector<vector<vector<float>>> prior_tissue_vector;
// pairwise phylogenetic distance between etissues
//vector<vector<float>> tissue_hierarchical_pairwise;		--> changed to the new variables above; as we prepare the hierarchy as what we need them to be




// learning control parameters:
int iter_learn_out = 1;  // iteration across all tissues
//int iter_learn_in = 200;  // iteration across all samples from one tissue 			--> (Mar.8, 2016) this is probably too much (we used 3:40 for running only one tissue; maybe 50 is good enough, as the changing speed is much slower)
int iter_learn_in = 50;
int batch_size = 20;  // better be 20												--> (Jan.27) testing mode


// test different learning rate
//float rate_learner = 1.0;  // the learning rate; this doesn't work
//float rate_learner = 0.1;  // the learning rate; this doesn't work
//float rate_learner = 0.01;  // the learning rate; this doesn't work
//float rate_learner = 0.001;  // the learning rate; works!!!; bench#3
float rate_learner = 0.0001;  // the learning rate; works!!!; bench#4
//float rate_learner = 0.00001;  // the learning rate; works!!!; bench#5				--> (Jan.27) the latest one
//float rate_learner = 0.000001;  // the learning rate




//========================================
// the following is the debugging routine
//========================================
/*
Feb.3:
Finally we cleaned all the bugs (if there is no more).
I'm now working on tuning the learn_rate parameter. It looks like large learn_rate will lead to the program go wild:
###########################
try different learning rate parameters:
2(1): all the parameters tend to be quite large, and there appears “Nan” at iter#8 in snp_cellenv
3(0.01): iter#29 “Nan” in snp_cellenv
4(0.0001): no “Nan”, and all the parameter values seem to be normal
5(0.000001): no “Nan”, and all the parameter values seem to be normal
###########################
"0.001" and "0.0001" seem good, yet it looks like they are still wilding the program, with different speeds.
I will try these two parameters with more iterations, to see where do they end up
###########################
I have run 200 iterations (1h12min) for 0.001 and 0.0001, and the results are in workbench#2 (0.001) and workbench#3 (0.0001)
I will run 1000 iterations (6h) for 0.001 and 0.0001, to see how do they converge; they are in workbench#4 and workbench#5
...
...
...
###########################



Feb.4:
something I planned to do, but finally give up as there is no need to do that:
1. add the stochastic module (the current method won't affect too much);
2. add errors to the true parameters (however, even we are in the true parameters, the model seem to diverse, so no meaning to do that);
3. learning several iterations in training set, predict on the testing set (directly predict other than saving the parameters first) (however, as the learning is diverging, this seems not so meaningful)



Feb.8 (Feb.12):
I will test the following this week:
(done) 1. add errors to the parameters (N(0, 1), as all the parameters are drawn from N(0, 1), so errors with that magtitude is acceptable; we can also try other error magtitude later on);
2. tune the prior (sparsity) strength, to see whether we have a better converging trend;
3. output the likelihood for the tensor, with and without errors (on parameters) added, and check its trend; we can do this on either training set or testing set;
4. it's the time now to make the program truly stochastic (for gradient descent)


*/
//======================================================================================================







// TODO: we need to think more about this, say, how to use the data from other epigenomics projects
// load all the cis- snp prior information (tissue specific) from prepared file outside
// fill in the following: vector<vector<vector<float>>> prior_tissue_vector
void opt_snp_prior_load()
{

	for(int i=0; i<num_etissue; i++)
	{
		vector<vector<float>> matrix;
		prior_tissue_vector.push_back(matrix);
	}


	/*
	// TODO: there should always be prior information in the repo
	// TODO: in the simulating data, we don't have this prior, so temporarily stop this

	// get the eTissue-index map
	unordered_map<string, string> index_map;  // for temporary usage
	char filename[100] = "../prior.score.unpruned/prior_tissue_index.txt";
	FILE * file_in = fopen(filename, "r");
	if(file_in == NULL)
	{
		fputs("File error\n", stderr); exit (1);
	}
	int input_length = 1000;
	char input[input_length];
	while(fgets(input, input_length, file_in) != NULL)
	{
		trim(input);

		const char * sep = "\t";
		char * p;
		p = strtok(input, sep);
		string eTissue = p;

		int count = 0;
		while(p)
		{
			count++;
			if(count == 1)  // this is the eTissue
			{
				p = strtok(NULL, sep);
				continue;
			}
			if(count == 2)  // this is the index
			{
				string index = p;
				index_map[eTissue] = index;
				break;
			}
		}
	}
	fclose (file_in);

	// get the prior score for each eTissue, on all chromosomes
	for( auto it = index_map.begin(); it != index_map.end(); ++it )
	{
		string eTissue = it->first;
		string index = it->second;
		vector<vector<float>> vec;
		prior_tissue_rep[eTissue] = vec;

		int i;
		for(i=0; i<NUM_CHR; i++)
		{
			int chr = i+1;
			vector<float> vec;
			prior_tissue_rep[eTissue].push_back(vec);

			//======== get all SNPs with their snp_info (count, position) ========
			char filename[100] = "../prior.score.unpruned/etissue";
			char temp[10];
			StrToCharSeq(temp, index);
			strcat(filename, temp);
			strcat(filename, "/chr");
			sprintf(temp, "%d", chr);
			strcat(filename, temp);
			strcat(filename, ".score");
			//puts("the current file worked on is: ");
			//puts(filename);

			FILE * file_in = fopen(filename, "r");
			if(file_in == NULL)
			{
				fputs("File error\n", stderr); exit (1);
			}

			int input_length = 100;
			char input[input_length];
			while(fgets(input, input_length, file_in) != NULL)
			{
				trim(input);

				float prior = stof(input);
				prior_tissue_rep[eTissue][i].push_back(prior);
			}
			fclose(file_in);
			//======================================
		}
	}
	*/

}





// load the pairwise tissue hierarchy from prepared file outside
// TODO: maybe we should check whether this makes the results better
void opt_tissue_hierarchy_load()
{

	/*	Mar.27: change the hierarchy data structures

	// target: vector<vector<float>> tissue_hierarchical_pairwise;
	// init
	for(int i=0; i<num_etissue; i++)
	{
		vector<float> vec;
		for(int j=0; j<num_etissue; j++)
		{
			vec.push_back(0);
		}
		tissue_hierarchical_pairwise.push_back(vec);
	}

	// load from data source
	char filename[100] = "../tissue_hierarchy_normalized.txt";
	FILE * file_in = fopen(filename, "r");
	if(file_in == NULL)
	{
		fputs("File error\n", stderr); exit (1);
	}
	int input_length = 100000;
	char input[input_length];
	while(fgets(input, input_length, file_in) != NULL)
	{
		trim(input);

		const char * sep = "\t";
		char * p;
		p = strtok(input, sep);
		string eTissue1 = p;
		int index1 = etissue_index_map[eTissue1];
		int index2 = 0;

		int count = 0;
		while(p)
		{
			count++;
			if(count == 1)  // this is the eTissue1
			{
				p = strtok(NULL, sep);
				continue;
			}
			if(count == 2)  // this is the eTissue2
			{
				string eTissue2 = p;
				int index2 = etissue_index_map[eTissue2];

				p = strtok(NULL, sep);
				continue;
			}
			if(count == 3)
			{
				float dist = stof(p);
				tissue_hierarchical_pairwise[index1][index2] = dist;
				tissue_hierarchical_pairwise[index2][index1] = dist;
				break;
			}
		}

	}
	fclose(file_in);


	*/



	//=====================================================
	//********************** hierarchy ********************
	//=====================================================

	cout << "now loading the tissue hierarchy..." << endl;



	// the hierarchy:
	// what to build for the hierarchy computing (prepared and renewed):
	//		[2]. hashing all the leaves to their parents (in order to retrieve the parental variable array)
	//		[4]. hashing all the internal nodes to their children and parent (with length to them, or the variance), in order to build the computational matrices
	//		[5]. having a bi-directional list for the internal nodes (in order to build and fill in computtional matrices)
	//		(6). building the tissue distance list (to its parent) based on etissue_list, to be used by the actual regularization
	
	//unordered_map<string, hierarchy_neighbor> hash_leaf_parent;		// --> [2]
	//unordered_map<string, vector< hierarchy_neighbor >> hash_internode_neighbor;
																		// --> [4]
	//vector<string> internode_list;									// --> [5]
	//unordered_map<string, int> internode_index_map;					// --> [5]
	//int num_internode;
	//vector<float> etissue_dis_par_list;								// --> (6)





	// TODO (Mar.29): load the following two:
	//unordered_map<string, hierarchy_neighbor> hash_leaf_parent;		// --> [2]
	//unordered_map<string, vector< hierarchy_neighbor >> hash_internode_neighbor;
																		// --> [4]
	while(1)
	{
		//
		//
		//
		//
		//

		string leaf;
		string parent;
		float branch;

		hierarchy_neighbor tuple;
		tuple.node = parent;
		tuple.branch = branch;
		hash_leaf_parent.emplace(leaf, tuple);
	}
	while(1)
	{
		//
		//
		//
		//
		//

		string internode;
		string neighbor1;		// child1 node
		float branch1;
		string neighbor2;		// child2 node
		float branch2;
		string neighbor3;		// parent node
		float branch3;

		vector<hierarchy_neighbor> vec;

		hierarchy_neighbor tuple1;
		tuple1.node = neighbor1;
		tuple1.branch = branch1;
		vec.push_back(tuple1);

		hierarchy_neighbor tuple2;
		tuple2.node = neighbor2;
		tuple2.branch = branch2;
		vec.push_back(tuple2);

		hierarchy_neighbor tuple3;
		tuple3.node = neighbor3;
		tuple3.branch = branch3;
		vec.push_back(tuple3);


		hash_internode_neighbor.emplace(internode, vec);
	}



	//vector<string> internode_list;									// --> [5]
	//unordered_map<string, int> internode_index_map;					// --> [5]
	//int num_internode;
	int count = 0;
	for( auto it = hash_internode_neighbor.begin(); it != hash_internode_neighbor.end(); ++it )
	{
		string internode = it->first;
		internode_list.push_back(internode);
		internode_index_map[internode] = count;
		count += 1;
	}
	num_internode = count;

	//vector<float> etissue_dis_par_list;								// --> (6)
	for(int i=0; i<num_etissue; i++)
	{
		etissue_dis_par_list.push_back(0);
	}
	for(auto it = hash_leaf_parent.begin(); it != hash_leaf_parent.end(); ++it )
	{
		string leaf = it->first;
		int etissue_index = etissue_index_map[leaf];
		etissue_dis_par_list[etissue_index] = (it->second).branch;
	}


	return;
}





void opt_para_init()
{
	puts("opt_para_init..");

	//=============== snp_dosage_list ===============
	for(int i=0; i<NUM_CHR; i++)
	{
		long num_temp = snp_name_list[i].size();
		float * p = (float *)calloc( num_temp, sizeof(float) );
		snp_dosage_list[i] = p;
	}

	//=============== gene_rpkm_exp ===============
	gene_rpkm_exp = (float *)calloc( num_gene, sizeof(float) );

	//=============== cellenv_hidden_var ===============
	cellenv_hidden_var = (float *)calloc( num_cellenv, sizeof(float) );

	//=============== batch_var ===============
	batch_var = (float *)calloc( num_batch, sizeof(float) );

	//=============== batch_hidden_var ===============
	batch_hidden_var = (float *)calloc( num_batch_hidden, sizeof(float) );



	//=============== cube_para_dev_cis_gene ===============
	for(int j=0; j<num_etissue; j++)
	{
		Matrix_imcomp matrix_imcomp;
		matrix_imcomp.init(num_gene);
		for(long int i=0; i<num_gene; i++)
		{
			string gene = gene_list[i];
			unordered_map<string, int>::const_iterator got = gene_xymt_rep.find(gene);
			if ( got != gene_xymt_rep.end() )
			{
				continue;
			}
			else
			{
				long int first = gene_cis_index[gene].first;  // index
				long int second = gene_cis_index[gene].second;  // index
				long int amount = second - first + 1;
				matrix_imcomp.init_element(i, amount + 1);

				// assing the chr and the tss:
				matrix_imcomp.init_assign_chr(i, gene_tss[gene].chr);
				//matrix_imcomp.init_assign_sst(i, gene_tss[gene].tss);		// Here is a BUG: sst != tss
																			// sst: the start index of cis SNPs for one gene; tss: transcription start site
				matrix_imcomp.init_assign_sst(i, gene_cis_index[gene].first);		// Here is a BUG: sst != tss
			}
		}
		cube_para_dev_cis_gene.push_back(matrix_imcomp);
	}

	//=============== matrix_para_dev_snp_cellenv ===============
	matrix_para_dev_snp_cellenv.init(num_cellenv, num_snp + 1);		// we do have the intercept term here

	//=============== cube_para_dev_cellenv_gene ===============
	for(int j=0; j<num_etissue; j++)
	{
		Matrix matrix;
		matrix.init(num_gene, num_cellenv + 1);						// we do have the intercept term here
		cube_para_dev_cellenv_gene.push_back(matrix);
	}

	//=============== matrix_para_dev_batch_batch_hidden ===============
	matrix_para_dev_batch_batch_hidden.init(num_batch_hidden, num_batch + 1);

	//=============== matrix_para_dev_batch_hidden_gene ===============
	matrix_para_dev_batch_hidden_gene.init(num_gene, num_batch_hidden + 1);



	//=====================================================
	//********************** hierarchy ********************
	//=====================================================
	cout << "initializing the hierarchy prior containers" << endl;

	//=============== cube_para_cis_gene_parent ===============
	for(int j=0; j<num_etissue; j++)
	{
		Matrix_imcomp matrix_imcomp;
		matrix_imcomp.init(num_gene);
		for(long int i=0; i<num_gene; i++)
		{
			int dimension = cube_para_dev_cis_gene[j].get_dimension2(i);
			matrix_imcomp.init_element(i, dimension);
		}
		cube_para_cis_gene_parent.push_back(matrix_imcomp);
	}

	//=============== cube_para_cellenv_gene_parent ===============
	for(int j=0; j<num_etissue; j++)
	{
		Matrix matrix;
		matrix.init(num_gene, num_cellenv + 1);						// we do have the intercept term here
		cube_para_cellenv_gene_parent.push_back(matrix);
	}


}




void opt_para_release()
{
	//=============== snp_dosage_list ===============
	for(int i=0; i<NUM_CHR; i++)
	{
		free(snp_dosage_list[i]);
	}

	//=============== gene_rpkm_exp ===============
	free(gene_rpkm_exp);

	//=============== cellenv_hidden_var ===============
	free(cellenv_hidden_var);

	//=============== batch_var ===============
	free(batch_var);

	//=============== batch_hidden_var ===============
	free(batch_hidden_var);



	//=============== cube_para_dev_cis_gene ===============
	for(int j=0; j<num_etissue; j++)
	{
		cube_para_dev_cis_gene[j].release();
	}


	//=============== matrix_para_dev_snp_cellenv ===============
	matrix_para_dev_snp_cellenv.release();


	//=============== cube_para_dev_cellenv_gene ===============
	for(int j=0; j<num_etissue; j++)
	{
		cube_para_dev_cellenv_gene[j].release();
	}


	//=============== matrix_para_dev_batch_batch_hidden ===============
	matrix_para_dev_batch_batch_hidden.release();


	//=============== matrix_para_dev_batch_hidden_gene ===============
	matrix_para_dev_batch_hidden_gene.release();


	//=====================================================
	//********************** hierarchy ********************
	//=====================================================
	cout << "releasing the hierarchy prior containers" << endl;

	//=============== cube_para_cis_gene_parent ===============
	for(int j=0; j<num_etissue; j++)
	{
		cube_para_cis_gene_parent[j].release();
	}

	//=============== cube_para_cellenv_gene_parent ===============
	for(int j=0; j<num_etissue; j++)
	{
		cube_para_cellenv_gene_parent[j].release();
	}


}




void GPU_init()
{
	int deviceID = 0;
    checkCudaErrors(hipSetDevice(deviceID));

	//=====================================================
	//================ GPU memory build-up ================
	//=====================================================
	//==== para_dev
	//matrix_para_dev_snp_cellenv --> float * d_para_dev_snp_cellenv
	long int dimension1 = matrix_para_dev_snp_cellenv.get_dimension1();
	long int dimension2 = matrix_para_dev_snp_cellenv.get_dimension2();
    //float * d_para_dev_snp_cellenv;			// whenever it comes to d_xxx (GPU device memory), it's an array other than matrix
    checkCudaErrors(hipMalloc(&d_para_dev_snp_cellenv, (dimension1*dimension2)*sizeof(float)));
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_dev_snp_cellenv.get_list(i);
    	long int pos_start = i * dimension2;
    	checkCudaErrors(hipMemcpy( (d_para_dev_snp_cellenv + pos_start), x, dimension2*sizeof(float), hipMemcpyHostToDevice));
    }
	//vector<Matrix> cube_para_dev_cellenv_gene --> vector<float *> d_list_para_dev_cellenv_gene
	dimension1 = cube_para_dev_cellenv_gene[0].get_dimension1();
	dimension2 = cube_para_dev_cellenv_gene[0].get_dimension2();
    //vector<float *> d_list_para_dev_cellenv_gene;			// whenever it comes to d_xxx (GPU device memory), it's an array other than matrix
    for(int j=0; j<num_etissue; j++)
    {
		float * d_para_dev_cellenv_gene;
	    checkCudaErrors(hipMalloc(&d_para_dev_cellenv_gene, (dimension1*dimension2)*sizeof(float)));
	    for(long int i=0; i<dimension1; i++)
	    {
	    	float * x = cube_para_dev_cellenv_gene[j].get_list(i);
	    	long int pos_start = i * dimension2;
	    	checkCudaErrors(hipMemcpy( (d_para_dev_cellenv_gene + pos_start), x, dimension2*sizeof(float), hipMemcpyHostToDevice));
	    }
	    d_list_para_dev_cellenv_gene.push_back(d_para_dev_cellenv_gene);
    }
	//Matrix matrix_para_dev_batch_batch_hidden --> float * d_para_dev_batch_batch_hidden
	dimension1 = matrix_para_dev_batch_batch_hidden.get_dimension1();
	dimension2 = matrix_para_dev_batch_batch_hidden.get_dimension2();
    //float * d_para_dev_batch_batch_hidden;			// whenever it comes to d_xxx (GPU device memory), it's an array other than matrix
    checkCudaErrors(hipMalloc(&d_para_dev_batch_batch_hidden, (dimension1*dimension2)*sizeof(float)));
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_dev_batch_batch_hidden.get_list(i);
    	long int pos_start = i * dimension2;
    	checkCudaErrors(hipMemcpy( (d_para_dev_batch_batch_hidden + pos_start), x, dimension2*sizeof(float), hipMemcpyHostToDevice));
    }
	//Matrix matrix_para_dev_batch_hidden_gene --> float * d_para_dev_batch_hidden_gene
	dimension1 = matrix_para_dev_batch_hidden_gene.get_dimension1();
	dimension2 = matrix_para_dev_batch_hidden_gene.get_dimension2();
    //float * d_para_dev_batch_hidden_gene;			// whenever it comes to d_xxx (GPU device memory), it's an array other than matrix
    checkCudaErrors(hipMalloc(&d_para_dev_batch_hidden_gene, (dimension1*dimension2)*sizeof(float)));
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_dev_batch_hidden_gene.get_list(i);
    	long int pos_start = i * dimension2;
    	checkCudaErrors(hipMemcpy( (d_para_dev_batch_hidden_gene + pos_start), x, dimension2*sizeof(float), hipMemcpyHostToDevice));
    }



    //==== para
	//matrix_para_snp_cellenv --> float * d_para_snp_cellenv
	dimension1 = matrix_para_snp_cellenv.get_dimension1();
	dimension2 = matrix_para_snp_cellenv.get_dimension2();
    //float * d_para_snp_cellenv;			// whenever it comes to d_xxx (GPU device memory), it's an array other than matrix
	checkCudaErrors(hipMalloc(&d_para_snp_cellenv, (dimension1*dimension2)*sizeof(float)));
	for(long int i=0; i<dimension1; i++)
	{
		float * x = matrix_para_snp_cellenv.get_list(i);
		long int pos_start = i * dimension2;
		checkCudaErrors(hipMemcpy( (d_para_snp_cellenv + pos_start), x, dimension2*sizeof(float), hipMemcpyHostToDevice));
    }
	//vector<Matrix> cube_para_cellenv_gene --> vector<float *> d_list_para_cellenv_gene
	dimension1 = cube_para_cellenv_gene[0].get_dimension1();
	dimension2 = cube_para_cellenv_gene[0].get_dimension2();
    //vector<float *> d_list_para_cellenv_gene;			// whenever it comes to d_xxx (GPU device memory), it's an array other than matrix
    for(int j=0; j<num_etissue; j++)
    {
		float * d_para_cellenv_gene;
	    checkCudaErrors(hipMalloc(&d_para_cellenv_gene, (dimension1*dimension2)*sizeof(float)));
	    for(long int i=0; i<dimension1; i++)
	    {
	    	float * x = cube_para_cellenv_gene[j].get_list(i);
	    	long int pos_start = i * dimension2;
	    	checkCudaErrors(hipMemcpy( (d_para_cellenv_gene + pos_start), x, dimension2*sizeof(float), hipMemcpyHostToDevice));
	    }
	    d_list_para_cellenv_gene.push_back(d_para_cellenv_gene);
    }
	//Matrix matrix_para_batch_batch_hidden --> float * d_para_batch_batch_hidden
	dimension1 = matrix_para_batch_batch_hidden.get_dimension1();
	dimension2 = matrix_para_batch_batch_hidden.get_dimension2();
    //float * d_para_batch_batch_hidden;			// whenever it comes to d_xxx (GPU device memory), it's an array other than matrix
    checkCudaErrors(hipMalloc(&d_para_batch_batch_hidden, (dimension1*dimension2)*sizeof(float)));
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_batch_batch_hidden.get_list(i);
    	long int pos_start = i * dimension2;
    	checkCudaErrors(hipMemcpy( (d_para_batch_batch_hidden + pos_start), x, dimension2*sizeof(float), hipMemcpyHostToDevice));
    }
	//Matrix matrix_para_batch_hidden_gene --> float * d_para_batch_hidden_gene
	dimension1 = matrix_para_batch_hidden_gene.get_dimension1();
	dimension2 = matrix_para_batch_hidden_gene.get_dimension2();
    //float * d_para_batch_hidden_gene;			// whenever it comes to d_xxx (GPU device memory), it's an array other than matrix
    checkCudaErrors(hipMalloc(&d_para_batch_hidden_gene, (dimension1*dimension2)*sizeof(float)));
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_batch_hidden_gene.get_list(i);
    	long int pos_start = i * dimension2;
    	checkCudaErrors(hipMemcpy( (d_para_batch_hidden_gene + pos_start), x, dimension2*sizeof(float), hipMemcpyHostToDevice));
    }



    //==== temp
    // temp space for Matrix Multiplication
	//float * d_temp_snp_cellenv;
	dimension1 = matrix_para_snp_cellenv.get_dimension1();
	dimension2 = matrix_para_snp_cellenv.get_dimension2();
	checkCudaErrors(hipMalloc(&d_temp_snp_cellenv, (dimension1*dimension2)*sizeof(float)));

	//float * d_temp_cellenv_gene;
	dimension1 = cube_para_cellenv_gene[0].get_dimension1();
	dimension2 = cube_para_cellenv_gene[0].get_dimension2();
	checkCudaErrors(hipMalloc(&d_temp_cellenv_gene, (dimension1*dimension2)*sizeof(float)));

	//float * d_temp_batch_batch_hidden;
	dimension1 = matrix_para_batch_batch_hidden.get_dimension1();
	dimension2 = matrix_para_batch_batch_hidden.get_dimension2();
    checkCudaErrors(hipMalloc(&d_temp_batch_batch_hidden, (dimension1*dimension2)*sizeof(float)));

	//float * d_temp_batch_hidden_gene;
	dimension1 = matrix_para_batch_hidden_gene.get_dimension1();
	dimension2 = matrix_para_batch_hidden_gene.get_dimension2();
    checkCudaErrors(hipMalloc(&d_temp_batch_hidden_gene, (dimension1*dimension2)*sizeof(float)));





    //==== temp (intermediate) variables
	checkCudaErrors(hipMalloc(&d_etissue_index_p, 1*sizeof(int)));

	//==== float * d_snp
	checkCudaErrors(hipMalloc(&d_snp, num_snp*sizeof(float)));

	//==== float * d_expr
	checkCudaErrors(hipMalloc(&d_expr, num_gene*sizeof(float)));

	//==== float * d_gene_rpkm_exp
	checkCudaErrors(hipMalloc(&d_gene_rpkm_exp, num_gene*sizeof(float)));

	//==== float * d_gene_rpkm_exp_cellenv
	checkCudaErrors(hipMalloc(&d_gene_rpkm_exp_cellenv, num_gene*sizeof(float)));

	//==== float * d_gene_rpkm_exp_batch
	checkCudaErrors(hipMalloc(&d_gene_rpkm_exp_batch, num_gene*sizeof(float)));

	//==== float * d_error_list
	checkCudaErrors(hipMalloc(&d_error_list, num_gene*sizeof(float)));

	//==== float * d_cellenv_hidden_var
	checkCudaErrors(hipMalloc(&d_cellenv_hidden_var, num_cellenv*sizeof(float)));

	//==== float * d_batch_var
	checkCudaErrors(hipMalloc(&d_batch_var, num_batch*sizeof(float)));

	//==== float * d_batch_hidden_var
	checkCudaErrors(hipMalloc(&d_batch_hidden_var, num_batch_hidden*sizeof(float)));

}




void GPU_release()
{

	//====================================================
	//================ GPU data retrieval ================
	//====================================================
	//==== para_dev
	//matrix_para_dev_snp_cellenv --> float * d_para_dev_snp_cellenv
	long int dimension1 = matrix_para_dev_snp_cellenv.get_dimension1();
	long int dimension2 = matrix_para_dev_snp_cellenv.get_dimension2();
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_dev_snp_cellenv.get_list(i);
    	long int pos_start = i * dimension2;
		checkCudaErrors(hipMemcpy(x, (d_para_dev_snp_cellenv + pos_start), dimension2*sizeof(float), hipMemcpyDeviceToHost));
    }
    checkCudaErrors(hipFree(d_para_dev_snp_cellenv));
	//vector<Matrix> cube_para_dev_cellenv_gene --> <float *> d_list_para_dev_cellenv_gene
	dimension1 = cube_para_dev_cellenv_gene[0].get_dimension1();
	dimension2 = cube_para_dev_cellenv_gene[0].get_dimension2();
	for(int j=0; j<num_etissue; j++)
	{
		float * d_para_dev_cellenv_gene = d_list_para_dev_cellenv_gene[j];
	    for(long int i=0; i<dimension1; i++)
	    {
	    	float * x = cube_para_dev_cellenv_gene[j].get_list(i);
	    	long int pos_start = i * dimension2;
			checkCudaErrors(hipMemcpy(x, (d_para_dev_cellenv_gene + pos_start), dimension2*sizeof(float), hipMemcpyDeviceToHost));
	    }
	    checkCudaErrors(hipFree(d_para_dev_cellenv_gene));
	}
	//Matrix matrix_para_dev_batch_batch_hidden --> float * d_para_dev_batch_batch_hidden
	dimension1 = matrix_para_dev_batch_batch_hidden.get_dimension1();
	dimension2 = matrix_para_dev_batch_batch_hidden.get_dimension2();
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_dev_batch_batch_hidden.get_list(i);
    	long int pos_start = i * dimension2;
		checkCudaErrors(hipMemcpy(x, (d_para_dev_batch_batch_hidden + pos_start), dimension2*sizeof(float), hipMemcpyDeviceToHost));
    }
    checkCudaErrors(hipFree(d_para_dev_batch_batch_hidden));
	//Matrix matrix_para_dev_batch_hidden_gene --> float * d_para_dev_batch_hidden_gene
	dimension1 = matrix_para_dev_batch_hidden_gene.get_dimension1();
	dimension2 = matrix_para_dev_batch_hidden_gene.get_dimension2();
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_dev_batch_hidden_gene.get_list(i);
    	long int pos_start = i * dimension2;
		checkCudaErrors(hipMemcpy(x, (d_para_dev_batch_hidden_gene + pos_start), dimension2*sizeof(float), hipMemcpyDeviceToHost));
    }
    checkCudaErrors(hipFree(d_para_dev_batch_hidden_gene));


    //==== para
	//matrix_para_snp_cellenv --> float * d_para_snp_cellenv
	dimension1 = matrix_para_snp_cellenv.get_dimension1();
	dimension2 = matrix_para_snp_cellenv.get_dimension2();
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_snp_cellenv.get_list(i);
    	long int pos_start = i * dimension2;
		checkCudaErrors(hipMemcpy(x, (d_para_snp_cellenv + pos_start), dimension2*sizeof(float), hipMemcpyDeviceToHost));
    }
    checkCudaErrors(hipFree(d_para_snp_cellenv));
	//vector<Matrix> cube_para_cellenv_gene --> <float *> d_list_para_cellenv_gene
	dimension1 = cube_para_cellenv_gene[0].get_dimension1();
	dimension2 = cube_para_cellenv_gene[0].get_dimension2();
	for(int j=0; j<num_etissue; j++)
	{
		float * d_para_cellenv_gene = d_list_para_cellenv_gene[j];
	    for(long int i=0; i<dimension1; i++)
	    {
	    	float * x = cube_para_cellenv_gene[j].get_list(i);
	    	long int pos_start = i * dimension2;
			checkCudaErrors(hipMemcpy(x, (d_para_cellenv_gene + pos_start), dimension2*sizeof(float), hipMemcpyDeviceToHost));
	    }
	    checkCudaErrors(hipFree(d_para_cellenv_gene));
	}
	//Matrix matrix_para_batch_batch_hidden --> float * d_para_batch_batch_hidden
	dimension1 = matrix_para_batch_batch_hidden.get_dimension1();
	dimension2 = matrix_para_batch_batch_hidden.get_dimension2();
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_batch_batch_hidden.get_list(i);
    	long int pos_start = i * dimension2;
		checkCudaErrors(hipMemcpy(x, (d_para_batch_batch_hidden + pos_start), dimension2*sizeof(float), hipMemcpyDeviceToHost));
    }
    checkCudaErrors(hipFree(d_para_batch_batch_hidden));
	//Matrix matrix_para_batch_hidden_gene --> float * d_para_batch_hidden_gene
	dimension1 = matrix_para_batch_hidden_gene.get_dimension1();
	dimension2 = matrix_para_batch_hidden_gene.get_dimension2();
    for(long int i=0; i<dimension1; i++)
    {
    	float * x = matrix_para_batch_hidden_gene.get_list(i);
    	long int pos_start = i * dimension2;
		checkCudaErrors(hipMemcpy(x, (d_para_batch_hidden_gene + pos_start), dimension2*sizeof(float), hipMemcpyDeviceToHost));
    }
    checkCudaErrors(hipFree(d_para_batch_hidden_gene));




    //==== para temp
    // temp space for Matrix Multiplication
	//float * d_temp_snp_cellenv;
    checkCudaErrors(hipFree(d_temp_snp_cellenv));

	//float * d_temp_cellenv_gene;
    checkCudaErrors(hipFree(d_temp_cellenv_gene));

	//float * d_temp_batch_batch_hidden;
    checkCudaErrors(hipFree(d_temp_batch_batch_hidden));

	//float * d_temp_batch_hidden_gene;
    checkCudaErrors(hipFree(d_temp_batch_hidden_gene));




    //==== temp (intermediate) variables
    //==== int * d_etissue_index_p
    checkCudaErrors(hipFree(d_etissue_index_p));

	//==== float * d_snp
    checkCudaErrors(hipFree(d_snp));

	//==== float * d_expr
    checkCudaErrors(hipFree(d_expr));

	//==== float * d_gene_rpkm_exp
    checkCudaErrors(hipFree(d_gene_rpkm_exp));

	//==== float * d_gene_rpkm_exp_cellenv
    checkCudaErrors(hipFree(d_gene_rpkm_exp_cellenv));

    //==== float * d_gene_rpkm_exp_batch
    checkCudaErrors(hipFree(d_gene_rpkm_exp_batch));

    //==== float * d_error_list
    checkCudaErrors(hipFree(d_error_list));

	//==== float * d_cellenv_hidden_var
    checkCudaErrors(hipFree(d_cellenv_hidden_var));

	//==== float * d_batch_var
    checkCudaErrors(hipFree(d_batch_var));

	//==== float * d_batch_hidden_var
    checkCudaErrors(hipFree(d_batch_hidden_var));






    checkCudaErrors(hipDeviceReset());
}






//function: mini-batches gradient; gradient descent
void optimize()
{
	puts("============== entering the optimization routine...");




	// Mar.30 DEBUG
	// the hierarchy code has not yet been finished; so I comment this routine here
	//puts("[xx] loading the tissue hierarchy...");
	//opt_tissue_hierarchy_load();




	puts("[xx] initializing the parameter space in this optimization routine...");
	opt_para_init();
	puts("[xx] loading the prior information for cis- snps...");
	opt_snp_prior_load();




	// TODO:
	// to define and initialze the parent parameter space (only for cis- regulator, and for cellular regulator)
	// we can use the similar data structure with the original two:
	//		vector<Matrix_imcomp> cube_para_cis_gene;
	//		vector<Matrix> cube_para_cellenv_gene;
	// make the variables visible to the hierarchical clustering sub-routine





	//======== likelihood ========
	// save the loglikelihood along the way
	char filename[100] = "../result/loglike.txt";
	FILE * file_out_loglike = fopen(filename, "w+");
	if(file_out_loglike == NULL)
	{
	    fputs("File error\n", stderr); exit(1);
	}


	//======== testing error (predictive error) ========
	sprintf(filename, "%s", "../result/test_error.txt");
	FILE * file_out_testerror = fopen(filename, "w+");
	if(file_out_testerror == NULL)
	{
	    fputs("File error\n", stderr); exit(1);
	}





	//======== GPU global variable init
	GPU_init();





	//============== timing starts ================
    struct timeval time_start;
    struct timeval time_end;
    double diff;
    gettimeofday(&time_start, NULL);




	for(int count1=0; count1<iter_learn_out; count1++)  // one count1 is for iteration across all tissues
	{

		for(int count2=0; count2<num_etissue; count2++)  // one count2 is for one tissue
		{
			string etissue = etissue_list[count2];
			int num_esample = eQTL_tissue_rep[etissue].size();



			//======== likelihood ========
			// indicating the current tissue
			char buf[100];
			sprintf(buf, "%s\t", etissue.c_str());
			fwrite(buf, sizeof(char), strlen(buf), file_out_loglike);



			//======== testing error (predictive error) ========
			//char buf[100];
			sprintf(buf, "%s\t", etissue.c_str());
			fwrite(buf, sizeof(char), strlen(buf), file_out_testerror);





			// entering this tissue
			for(int count3=0; count3<iter_learn_in; count3++)  // one count3 is for a batch_size mini-batch in current tissue
			{


				//
				// TODO: change this module to the real stochastic one (other than rounding over all the samples)
				//
				// QUESTION: can we shuffle the sample list?
				//
				// ANS: not urgent to do that, as the current setting is perceptron
				//


				int pos_start = (batch_size * count3) % (num_esample);
				printf("[@@@] now we are working on %d iter_out (%d total), eTissue #%d (%d total) -- %s (%d training samples in), #%d mini-batch (%d batch size, rounding all samples).\n", count1+1, iter_learn_out, count2+1, num_etissue, etissue.c_str(), num_esample, count3+1, batch_size);
				forward_backward_prop_batch(etissue, pos_start, num_esample);
				// leaving this mini-batch



				// // DEBUG:
				// // we do only one sample (or one mini-batch) for the current tissue
				// break;


				/*
				//======== parameter check ========
				// check "nan" after this mini-batch
				int flag = para_check_nan(etissue);
				if(flag == 1)
				{
					//
					cout << "we get nan..." << endl;
					cout << "the # of mini-batch we are in is ";
					cout << count3 + 1 << endl;
					break;
				}




				//=========================================================================================================
				//****************************************** loglike or testerror *****************************************
				//=========================================================================================================
				int num_check_every = 5;

				//======== likelihood ========
				// (Feb.14) after we finish this mini-batch, we'll need to check the log-likelihood of the model (for the current tissue); or maybe check every several mini-batches
				if(count3 % num_check_every == 0)
				{
					float loglike;
					loglike = cal_loglike(etissue);

					char buf[1024];
					sprintf(buf, "%f\t", loglike);
					fwrite(buf, sizeof(char), strlen(buf), file_out_loglike);
				}

				//======== testing error (predictive error) ========
				// we can check every several mini-batches
				if(count3 % num_check_every == 0)
				{
					float testerror;
					testerror = cal_testerror(etissue);

					//char buf[1024];
					sprintf(buf, "%f\t", testerror);
					fwrite(buf, sizeof(char), strlen(buf), file_out_testerror);
				}
				*/





				// DEBUG
				// Mar.30: DEBUG: run only mini-batch to see the functionality
				break;






			}
			// leaving this etissue


			//======== likelihood ========
			// finish this line in the likelihood file
			fwrite("\n", sizeof(char), 1, file_out_loglike);


			//======== testing error (predictive error) ========
			fwrite("\n", sizeof(char), 1, file_out_testerror);






			// DEBUG: won't consider other tissues
			break;







			// DEBUG: we do think all the tissues
			//DEBUG: won't consider other tissues; only consider the current tissue
			//break;



		}
		//
		// whenever we finish one iteration across all tissues, we should save the learned parameters
		//
		//para_inter_save(count1);
		//
		//


		// (Mar.22, 2016) TODO: now we do the tissue hierarchical prior
		// we only care about the following two:
		//		vector<Matrix_imcomp> cube_para_cis_gene;
		//		vector<Matrix> cube_para_cellenv_gene;
		// we want do things more general




		//hierarchy();




	}// leave the current outer iteration




	//============== timing ends ================
	gettimeofday(&time_end, NULL);
	diff = (double)(time_end.tv_sec-time_start.tv_sec) + (double)(time_end.tv_usec-time_start.tv_usec)/1000000;
	printf("Time used totally is %f seconds.\n", diff);



	//======== GPU global variable release
	GPU_release();






	//======== likelihood ========
	// finish the likelihood file
	fclose(file_out_loglike);


	//======== testing error (predictive error) ========
	fclose(file_out_testerror);








	opt_para_release();
	puts("============== leaving the optimization routine...");
}


